#include "hip/hip_runtime.h"
#include "xnbhd_diff_impl_gpu.h"
#include <sstream>

__global__ void apply_differencing_impl(
    const float* input_tensor,
    float* output_tensor,
    long in_nk,
    long in_nr,
    long in_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    // Current GPU index
    long i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i > n) { return; }

    // Find neighborhood indices (center comparison pixel location)
    long nbhd_c = i/nbhd_nc % in_nc;               // also center column
    long nbhd_r = i/nbhd_nc/in_nc/nbhd_nr % in_nr; // also center row
    long k = i/nbhd_nc/in_nc/nbhd_nr/in_nr % in_nk;
    long sample = i/nbhd_nc/in_nc/nbhd_nr/in_nr/in_nk;

    // Find in-neighborhood row and column indices
    long in_nbhd_c = i % nbhd_nc;
    long in_nbhd_r = i/nbhd_nc/in_nc % nbhd_nr;

    // Find the "neighborhood image" row and column indices
    long in_c = nbhd_c - nbhd_nc/2 + in_nbhd_c;
    long in_r = nbhd_r - nbhd_nr/2 + in_nbhd_r;

    // `flag` flips which image in the image pair is the "neighborhood image".
    long flag = (sample % 2 == 0) ? 1 : -1;
    // If the neighborhood indexing exceeds the size of the "neighborhood image", set the difference output to 0
    // (no activation).
    if (in_c < 0 || in_r < 0 || in_nc <= in_c ||  in_nr <= in_r) {
        output_tensor[i] = 0.0;
    }
    else {
        long idx1 = ((sample*in_nk + k)*in_nr + nbhd_r)*in_nc + nbhd_c;
        long idx2 = (((sample+flag)*in_nk + k)*in_nr + in_r)*in_nc + in_c;
        output_tensor[i] = input_tensor[idx1]-input_tensor[idx2];
    }
}

__global__ void get_differencing_gradient_impl(
    const float* gradient_input,
    float* gradient_output,
    long out_nk,
    long out_nr,
    long out_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    // Current GPU index
    long i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i > n) { return; }

    // Find the output indices
    long out_c = i % out_nc;
    long out_r = i/out_nc % out_nr;
    long k = i/out_nc/out_nr % out_nk;
    long sample = i/out_nc/out_nr/out_nk;

    // Backpropagate gradients for when the current pixel was the center
    // comparison pixel.
    gradient_output[i] = 0;
    for (long r = out_r*nbhd_nr; r < (out_r+1)*nbhd_nr; ++r) {
        long offset = ((sample*out_nk + k)*out_nr*nbhd_nr + r)*out_nc*nbhd_nc;
        for (long c = out_c*nbhd_nc; c < (out_c+1)*nbhd_nc; ++c) {
            gradient_output[i] += gradient_input[offset + c];
        }
    }

    long flag = (sample % 2 == 0) ? 1 : -1;
    long r_off = nbhd_nr/2;
    long c_off = nbhd_nc/2;

    // Backpropagate gradients for when the current pixel was part of the
    // "neighborhood image"
    long out_nbhd_r = 0;  // in-neighborhood row index
    for (long r = out_r+r_off; r >= out_r-r_off; --r) {
        if (r < 0 || r >= out_nr) {
            ++out_nbhd_r;
            continue;
        }
        long out_nbhd_c = 0;  // in-neighborhood column index
        long offset = (((sample+flag)*out_nk + k)*out_nr*nbhd_nr + r*nbhd_nr + out_nbhd_r)*out_nc*nbhd_nc;
        ++out_nbhd_r;

        for (long c = out_c+c_off; c >= out_c-c_off; --c) {
            if (c < 0 || c >= out_nc) {
                ++out_nbhd_c;
                continue;
            }
            gradient_output[i] -= gradient_input[offset + c*nbhd_nc + out_nbhd_c];
            ++out_nbhd_c;
        }
    }
}

void launch_differencing_kernel(
    const float* input_tensor,
    float* data_output,
    long in_nk,
    long in_nr,
    long in_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    int nblocks;
    int nthreads;
    hipError_t err = hipOccupancyMaxPotentialBlockSize(&nblocks, &nthreads, apply_differencing_impl);
    if (err != hipSuccess) {
        std::ostringstream oss;
        oss << "CUDA error when calling launch_differencing_kernel"
            << "\ncode: " << err << ", error: " << hipGetErrorString(err);
        throw std::runtime_error(oss.str());
    }
    apply_differencing_impl<<<nblocks, nthreads>>>(input_tensor, data_output, in_nk, in_nr, in_nc, nbhd_nr, nbhd_nc, n);
}

void launch_differencing_gradient_kernel(
    const float* gradient_input,
    float* gradient_output,
    long in_nk,
    long in_nr,
    long in_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    int nblocks;
    int nthreads;
    hipError_t err = hipOccupancyMaxPotentialBlockSize(&nblocks, &nthreads, get_differencing_gradient_impl);
    if (err != hipSuccess) {
        std::ostringstream oss;
        oss << "CUDA error when calling launch_differencing_gradient_kernel"
            << "\ncode: " << err << ", error: " << hipGetErrorString(err);
        throw std::runtime_error(oss.str());
    }
    get_differencing_gradient_impl<<<nblocks, nthreads>>>(gradient_input, gradient_output,
                                                          in_nk, in_nr, in_nc,
                                                          nbhd_nr, nbhd_nc,
                                                          n);
}
