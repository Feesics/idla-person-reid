#include "hip/hip_runtime.h"
#include "difference_impl.h"

#include <dlib/dnn/cuda_utils.h>

__global__ void apply_differencing_impl(
    const float* input_tensor,
    float* output_tensor,
    long in_nk,
    long in_nr,
    long in_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    for (auto i : dlib::cuda::grid_stride_range(0, n))
    {
        // Find neighborhood indices
        long nbhd_c = i/nbhd_nc % in_nc;               // also center column
        long nbhd_r = i/nbhd_nc/in_nc/nbhd_nr % in_nr; // also center row
        long k = i/nbhd_nc/in_nc/nbhd_nr/in_nr % in_nk;
        long sample = i/nbhd_nc/in_nc/nbhd_nr/in_nr/in_nk;

        // Find in-neighborhood indices
        long in_nbhd_c = i % nbhd_nc;
        long in_nbhd_r = i/nbhd_nc/in_nc % nbhd_nr;

        // Find the second input tensor indices
        long in_c = nbhd_c - nbhd_nc/2 + in_nbhd_c;
        long in_r = nbhd_r - nbhd_nr/2 + in_nbhd_r;

        if (in_c <= 0 || in_r <= 0 || in_nc <= in_c ||  in_nr <= in_r) {
            output_tensor[2*i] = 0.0;
            output_tensor[2*i+1] = 0.0;
        }
        else {
            long idx1 = ((2*sample*in_nk + k)*in_nr + nbhd_r)*in_nc + nbhd_c;
            long idx2 = (((2*sample+1)*in_nk + k)*in_nr + in_r)*in_nc + in_c;
            output_tensor[2*i] = input_tensor[idx1]-input_tensor[idx2];
            output_tensor[2*i+1] = input_tensor[idx2]-input_tensor[idx1];
        }
    }
}

__global__ void get_differencing_gradient_impl(
    const float* gradient_input,
    float* gradient_output,
    long out_nk,
    long out_nr,
    long out_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    for (auto i : dlib::cuda::grid_stride_range(0, n))
    {
        // Find the output indices
        long out_c = i % out_nc;
        long out_r = i/out_nc % out_nr;
        long k = i/out_nc/out_nr % out_nk;
        long sample = i/out_nc/out_nr/out_nk;

        gradient_output[i] = 0;
        for (long r = out_r*nbhd_nr; r < (out_r+1)*nbhd_nr; ++r) {
            long offset = ((sample*out_nk + k)*out_nr*nbhd_nr + r)*out_nc*nbhd_nc;
            for (long c = out_c*nbhd_nc; c < (out_c+1)*nbhd_nc; ++c) {
                gradient_output[i] += gradient_input[offset + c];
            }
        }

        // Specify in-neighborhood indexes
        long out_nbhd_r = 0;
        long out_nbhd_c = 0;

        long flag = (sample % 2 == 0) ? 1 : -1;
        long r_off = nbhd_nr/2;
        for (long r = out_r+r_off; r > out_r-r_off; --r) {
            if (r < 0 || r >= out_nr) {
                ++out_nbhd_r;
                continue;
            }
            long offset = (((sample + flag)*out_nk + k)*out_nr*nbhd_nr + r*nbhd_nr + out_nbhd_r)*out_nc*nbhd_nc;
            long c_off = nbhd_nc/2;
            ++out_nbhd_r;

            for (long c = out_c+c_off; c > out_c-c_off; --c) {
                if (c < 0 || c >= out_nc) {
                    ++out_nbhd_c;
                    continue;
                }
                gradient_output[i] += -gradient_input[offset + c*nbhd_nc + out_nbhd_c];
                ++out_nbhd_c;
            }
        }
    }
}

void idla::launch_differencing_kernel(
    const float* input_tensor,
    float* data_output,
    long in_nk,
    long in_nr,
    long in_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    dlib::cuda::launch_kernel(apply_differencing_impl,
                              dlib::cuda::max_jobs(n),
                              input_tensor,
                              data_output,
                              in_nk, in_nr, in_nc,
                              nbhd_nr, nbhd_nc, n);
}

void idla::launch_differencing_gradient_kernel(
    const float* gradient_input,
    float* gradient_output,
    long in_nk,
    long in_nr,
    long in_nc,
    long nbhd_nr,
    long nbhd_nc,
    long n
)
{
    dlib::cuda::launch_kernel(get_differencing_gradient_impl,
                              dlib::cuda::max_jobs(n),
                              gradient_input,
                              gradient_output,
                              in_nk, in_nr, in_nc,
                              nbhd_nr, nbhd_nc, n);
}
